#include "hip/hip_runtime.h"
#include "prox/prox_moreau.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include "config.hpp"

template<typename T>
__global__
void MoreauPrescale(T *d_scaled_arg,
                    T *d_arg,
                    T *d_tau,
                    T tau,
                    size_t count,
                    bool invert_tau)
{ 
  int tx = threadIdx.x + blockDim.x * blockIdx.x;

  if(tx < count) {
    if(invert_tau)
      d_scaled_arg[tx] = d_arg[tx] * (tau * d_tau[tx]);
    else
      d_scaled_arg[tx] = d_arg[tx] / (tau * d_tau[tx]);
  }
}

template<typename T>
__global__
void MoreauPostscale(T *d_result,
                     T *d_arg,
                     T *d_tau,
                     T tau,
                     size_t count,
                     bool invert_tau)
{ 
  int tx = threadIdx.x + blockDim.x * blockIdx.x;

  if(tx < count) {
    if(invert_tau)
      d_result[tx] = d_arg[tx] - d_result[tx] / (tau * d_tau[tx]);
    else
      d_result[tx] = d_arg[tx] - tau * d_tau[tx] * d_result[tx];
  }
}

template<typename T>
ProxMoreau<T>::ProxMoreau(Prox<T> *conjugate)
    : Prox<T>(*conjugate), conjugate_(conjugate) {
}

template<typename T>
ProxMoreau<T>::~ProxMoreau() {
  Release();
}

template<typename T>
bool ProxMoreau<T>::Init() {
  hipMalloc((void **)&d_scaled_arg_, sizeof(T) * (this->count_ * this->dim_));

  bool success = conjugate_->Init();
  
  return success && (hipGetLastError() == hipSuccess);
}

template<typename T>
void ProxMoreau<T>::Release() {
  hipFree(d_scaled_arg_);
}

template<typename T>
void ProxMoreau<T>::EvalLocal(T *d_arg,
                              T *d_res,
                              T *d_tau,
                              T tau,
                              bool invert_tau)
{
  size_t total_count = this->count_ * this->dim_;
  dim3 block(kBlockSizeCUDA, 1, 1);
  dim3 grid((total_count + block.x - 1) / block.x, 1, 1);

  // scale argument
  MoreauPrescale<T>
      <<<grid, block>>>(d_scaled_arg_,
                        d_arg,
                        d_tau,
                        tau,
                        total_count,
                        invert_tau);

  // compute prox with scaled argument
  conjugate_->EvalLocal(d_scaled_arg_, d_res, d_tau, tau, !invert_tau);

  // combine back to get result of conjugate prox
  MoreauPostscale<T>
      <<<grid, block>>>(d_res,
                        d_arg,
                        d_tau,
                        tau,
                        total_count,
                        invert_tau);


  //  conjugate_->EvalLocal(d_arg, d_res, d_tau, tau, false);
}

template<typename T>
size_t ProxMoreau<T>::gpu_mem_amount() {
  return this->count_ * this->dim_ * sizeof(T);
}


// Explicit template instantiation
template class ProxMoreau<float>;
template class ProxMoreau<double>;
