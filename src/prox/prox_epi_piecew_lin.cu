#include "hip/hip_runtime.h"
#include "prox/prox_epi_piecew_lin.hpp"

#include <cassert>
#include <hip/hip_runtime.h>
#include "config.hpp"
#include <iostream>

template<typename T>
__global__
void ProxEpiPiecewLinKernel(T *d_arg,
                            T *d_res,
                            EpiPiecewLinCoeffsDevice<T> coeffs,
                            size_t count,
                            bool interleaved)
{
  size_t tx = threadIdx.x + blockDim.x * blockIdx.x;

  if(tx < count) {
    T result[2];

    // get v = (x0, y0) and alpha,beta and count,index
    T alpha = coeffs.d_ptr_alpha[tx];
    T beta = coeffs.d_ptr_beta[tx];
    size_t count_local = coeffs.d_ptr_count[tx];
    size_t index = coeffs.d_ptr_index[tx];

    T v[2];
    if(interleaved) {
      v[0] = d_arg[tx * 2 + 0];
      v[1] = d_arg[tx * 2 + 1];
    } else {
      v[0] = d_arg[tx + count * 0];
      v[1] = d_arg[tx + count * 1];
    }
    
    // compute vector normal to slope for feasibility-check
    T n_slope[2];
    n_slope[0] = alpha;
    n_slope[1] = -1;
    
    T x1 = coeffs.d_ptr_x[index];
    T y1 = coeffs.d_ptr_y[index];
    T p[2];
    p[0] = x1;
    p[1] = y1;

    bool feasible_1 = PointInHalfspace(v, p, n_slope, 2);
    
    T n_halfspace[2];
    n_halfspace[0] = 1;
    n_halfspace[1] = alpha;

    bool halfspace_1 = PointInHalfspace(v, p, n_halfspace, 2);

    bool projected = false;

    if(!feasible_1 && halfspace_1) {
      // point is not feasible wrt to 0-th piece and
      //  lies in rectangle => projection is the 
      //  respective half space projection

      T t = x1*n_slope[0] + y1*n_slope[1];
      ProjectHalfspace(v, n_slope, t, result, 2);
      projected = true;
    }

    if(!projected) {
      for(size_t i = 0; i < count_local-1; i++) {
        // read "knick" at i+1
        T x2 = coeffs.d_ptr_x[index+i+1];
        T y2 = coeffs.d_ptr_y[index+i+1];

        // compute slope
        T c = (y2-y1) / (x2-x1);

        // compute vector normal to slope
        n_slope[0] = c;
        n_slope[1] = -1;

        // check whether point v is feasible wrt i-th piece
        bool feasible_2 = PointInHalfspace(v, p, n_slope, 2);

        n_halfspace[0] = -1;
        n_halfspace[1] = -c;


        bool halfspace_2 = PointInHalfspace(v, p, n_halfspace, 2);

        p[0] = x2;
        p[1] = y2;
        if(!feasible_1 || !feasible_2) {
          // point is not feasible wrt to i-th piece or (i-1)-th piece
          if(!halfspace_1 && !halfspace_2) {
            // point lies in (i-1)-th normal cone => projection is the "knick"
            result[0] = x1;
            result[1] = y1; 

            projected = true;
            break;
          }

          // compute inverse normal -n s.t. the two normals n and -n
          //  together with the two knicks define a reactangle
          n_halfspace[0] = -n_halfspace[0];
          n_halfspace[1] = -n_halfspace[1];

          // check wether point lies in i-th halfspace
          halfspace_1 = PointInHalfspace(v, p, n_halfspace, 2);
          if(halfspace_2 && halfspace_1) {
            // point lies in i-th rectangle => projection is the 
            //  respective half space projection

            T t = x1*n_slope[0] + y1*n_slope[1];
            ProjectHalfspace(v, n_slope, t, result, 2);

            projected = true;
            break;
          }

        }

        // hand over variables for next iteration
        x1 = x2;
        y1 = y2;
        feasible_1 = feasible_2;
      }
    }

 

    if(!projected) {
      // compute vector normal to slope
      n_slope[0] = beta;
      n_slope[1] = -1; 

      // check whether point v is feasible wrt i-th piece
      bool feasible_2 = PointInHalfspace(v, p, n_slope, 2);

      n_halfspace[0] = -1;
      n_halfspace[1] = -beta;

      bool halfspace_2 = PointInHalfspace(v, p, n_halfspace, 2);

      if(!feasible_1 || !feasible_2) {
        // point is not feasible wrt to i-th piece or (i-1)-th piece
        if(!halfspace_1 && !halfspace_2) {
          // point lies in last normal cone => projection is the last "knick"
          result[0] = x1;
          result[1] = y1; 

          projected = true;
        } else if(halfspace_2) {
          // point lies in last rectangle => projection is the 
          //  respective half space projection

          T t = x1*n_slope[0] + y1*n_slope[1];
          ProjectHalfspace(v, n_slope, t, result, 2);

          projected = true;
        }
      }
    }

    // point has not been projected. That means we output the original point    
    if(!projected) {
      result[0] = v[0];
      result[1] = v[1];      
    }
    
    // write out result
    if(interleaved) {
      d_res[tx * 2 + 0] = result[0];
      d_res[tx * 2 + 1] = result[1];
    } else {
      d_res[tx + count * 0] = result[0];
      d_res[tx + count * 1] = result[1];
    }
  }
}

template<typename T>
ProxEpiPiecewLin<T>::ProxEpiPiecewLin(size_t index,
                                      size_t count,
                                      bool interleaved,
                                      const EpiPiecewLinCoeffs<T>& coeffs)
    
    : Prox<T>(index, count, 2, interleaved, false), coeffs_(coeffs)
{
}

template<typename T>
ProxEpiPiecewLin<T>::~ProxEpiPiecewLin() {
  Release();
}

template<typename T>
bool ProxEpiPiecewLin<T>::Init() {
  
  if(coeffs_.x.empty() || coeffs_.y.empty() 
    || coeffs_.alpha.empty() || coeffs_.beta.empty() || 
       coeffs_.index.empty() || coeffs_.count.empty())
    return false;
  

  T *d_ptr_T = NULL;

  // copy x and y
  size_t count_xy = coeffs_.index[this->count_-1] + coeffs_.count[this->count_-1];

  size_t size = count_xy * sizeof(T);

  // copy x
  hipMalloc((void **)&d_ptr_T, size);
  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  hipMemcpy(d_ptr_T, &coeffs_.x[0], size, hipMemcpyHostToDevice);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  coeffs_dev_.d_ptr_x = d_ptr_T;

  // copy y
  hipMalloc((void **)&d_ptr_T, size);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }

  hipMemcpy(d_ptr_T, &coeffs_.y[0], size, hipMemcpyHostToDevice);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  coeffs_dev_.d_ptr_y = d_ptr_T;

  // copy alpha
  size = this->count_ * sizeof(T);
  hipMalloc((void **)&d_ptr_T, size);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  hipMemcpy(d_ptr_T, &coeffs_.alpha[0], size, hipMemcpyHostToDevice);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  coeffs_dev_.d_ptr_alpha = d_ptr_T;


  // copy beta
  hipMalloc((void **)&d_ptr_T, size);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }

  hipMemcpy(d_ptr_T, &coeffs_.beta[0], size, hipMemcpyHostToDevice);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  coeffs_dev_.d_ptr_beta = d_ptr_T;


  // copy count
  size = this->count_ * sizeof(size_t);

  size_t *d_ptr_size_t = NULL;
  hipMalloc((void **)&d_ptr_size_t, size);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }

  hipMemcpy(d_ptr_size_t, &coeffs_.count[0], size, hipMemcpyHostToDevice);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  coeffs_dev_.d_ptr_count = d_ptr_size_t;

  // copy index
  hipMalloc((void **)&d_ptr_size_t, size);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }

  hipMemcpy(d_ptr_size_t, &coeffs_.index[0], size, hipMemcpyHostToDevice);
  err = hipGetLastError();
  if(err != hipSuccess) {
    std::cout << hipGetErrorString(err)<< std::endl;            
    return false;
  }
  coeffs_dev_.d_ptr_index = d_ptr_size_t;

  return true;
}

template<typename T>
void ProxEpiPiecewLin<T>::Release() {
  hipFree(coeffs_dev_.d_ptr_x);
  hipFree(coeffs_dev_.d_ptr_y);
  hipFree(coeffs_dev_.d_ptr_alpha);
  hipFree(coeffs_dev_.d_ptr_beta);
  hipFree(coeffs_dev_.d_ptr_index);
  hipFree(coeffs_dev_.d_ptr_count);

}

template<typename T>
void ProxEpiPiecewLin<T>::EvalLocal(T *d_arg,
                                    T *d_res,
                                    T *d_tau,
                                    T tau,
                                    bool invert_tau)
{
  dim3 block(kBlockSizeCUDA, 1, 1);
  dim3 grid((this->count_ + block.x - 1) / block.x, 1, 1);

   ProxEpiPiecewLinKernel<T>
      <<<grid, block>>>(
          d_arg,
          d_res,
          coeffs_dev_,
          this->count_,
          this->interleaved_);
}

// Explicit template instantiation
template class ProxEpiPiecewLin<float>;
template class ProxEpiPiecewLin<double>;
