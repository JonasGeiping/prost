#include "hip/hip_runtime.h"
/**
* This file is part of prost.
*
* Copyright 2016 Thomas Möllenhoff <thomas dot moellenhoff at in dot tum dot de> 
* and Emanuel Laude <emanuel dot laude at in dot tum dot de> (Technical University of Munich)
*
* prost is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* prost is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with prost. If not, see <http://www.gnu.org/licenses/>.
*/

#include "prost/prox/prox.hpp"
#include <ctime>

namespace prost {

template<typename T>
void Prox<T>::Eval(
  thrust::device_vector<T>& result, 
  const thrust::device_vector<T>& arg, 
  const thrust::device_vector<T>& tau_diag, 
  T tau,
  bool invert_tau)
{
  EvalLocal(
    result.begin() + index_,
    result.begin() + index_ + size_,
    arg.cbegin() + index_,
    arg.cbegin() + index_ + size_,
    tau_diag.cbegin() + index_,
    tau_diag.cbegin() + index_ + size_,
    tau,
    invert_tau);
}

template<typename T>
double Prox<T>::Eval(
  std::vector<T>& result, 
  const std::vector<T>& arg, 
  const std::vector<T>& tau_diag, 
  T tau) 
{
  const int repeats = 1;

  const thrust::device_vector<T> d_arg(arg.begin(), arg.end());
  thrust::device_vector<T> d_res;
  d_res.resize(arg.size());
  const thrust::device_vector<T> d_tau(tau_diag.begin(), tau_diag.end());

  const clock_t begin_time = clock();
  for(int i = 0; i < repeats; i++)
  {
    Eval(d_res, d_arg, d_tau, tau);
    hipDeviceSynchronize();
  }
  double s = (double)(clock() - begin_time) / CLOCKS_PER_SEC;

  result.resize(arg.size());
  thrust::copy(d_res.begin(), d_res.end(), result.begin());

  return (s * 1000 / (double)repeats);
}

template <typename T>
void Prox<T>::get_separable_structure(
    vector<std::tuple<size_t, size_t, size_t> >& sep)
{
  sep.push_back( std::tuple<size_t, size_t, size_t> (index_, size_, 1) );
}


// Explicit template instantiation
template class Prox<float>;
template class Prox<double>;

} // namespace prost

