#include "util/util.hpp"

#include <string>
#include <iostream>

using namespace std;

// cuda error checking
string prev_file = "";
int prev_line = 0;
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        exit(1);
    }
    prev_file = file;
    prev_line = line;
}
