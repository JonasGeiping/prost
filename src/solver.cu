#include "hip/hip_runtime.h"
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <list>
#include <sstream>

#include "prost/solver.hpp"
#include "prost/backend/backend.hpp"
#include "prost/problem.hpp"

namespace prost {

using std::cout;
using std::endl;

template<typename T>
std::list<double> linspace(T start_in, T end_in, int num_in) {
  double start = static_cast<double>(start_in);
  double end = static_cast<double>(end_in);
  double num = static_cast<double>(num_in);
  double delta = (end - start) / (num - 1);

  std::list<double> linspaced; 
  for(int i = 0; i < num; ++i) 
    linspaced.push_back(start + delta * i);

  linspaced.push_back(end);
  
  return linspaced;
}

template<typename T>
Solver<T>::Solver(std::shared_ptr<Problem<T> > problem, std::shared_ptr<Backend<T> > backend) 
  : problem_(problem), backend_(backend)
{
}

template<typename T> 
void Solver<T>::SetOptions(const typename Solver<T>::Options& opts) {
  opts_ = opts;

  if(opts_.verbose)
  {
    // TODO: output individual solver options
  }
}

template<typename T>
void Solver<T>::SetStoppingCallback(const typename Solver<T>::StoppingCallback& cb) {
  stopping_cb_ = cb;
}

template<typename T>
void Solver<T>::SetIntermCallback(const typename Solver<T>::IntermCallback& cb) {
  interm_cb_ = cb;
}

template<typename T>
void Solver<T>::Initialize() {
  problem_->Initialize();

  backend_->SetProblem(problem_);
  backend_->SetOptions(opts_);
  backend_->Initialize();

  cur_primal_sol_.resize( problem_->ncols() );
  cur_dual_sol_.resize( problem_->nrows() );

  if(opts_.verbose) {
    size_t mem = problem_->gpu_mem_amount() + backend_->gpu_mem_amount();
    
    std::cout << "Initialized solver successfully. Problem dimension:" << std::endl;
    std::cout << "# primal variables: " << problem_->ncols() << std::endl;
    std::cout << "# dual variables: " << problem_->nrows() << std::endl;
    std::cout << "Memory requirements: " << mem / (1024 * 1024) << "MB." << std::endl;
  }
}

template<typename T>
typename Solver<T>::ConvergenceResult Solver<T>::Solve() {
  // iterations to display
  std::list<double> cb_iters =
      linspace(0, opts_.max_iters - 1, opts_.num_cback_calls);
  
  for(int i = 0; i < opts_.max_iters; i++) {    
    backend_->PerformIteration();

    // check if solver has converged
    T primal_res = backend_->primal_residual();
    T dual_res = backend_->dual_residual();
    T pv_norm = backend_->primal_var_norm();
    T dv_norm = backend_->dual_var_norm();

    bool is_converged = false;
    bool is_stopped = stopping_cb_();

    T eps_pri = std::sqrt(problem_->nrows()) * opts_.tol_abs_primal +
      opts_.tol_rel_primal * pv_norm;
    
    T eps_dua = std::sqrt(problem_->ncols()) * opts_.tol_abs_dual +
      opts_.tol_rel_dual * dv_norm;

    if((primal_res < eps_pri) && (dual_res < eps_dua))
      is_converged = true;

    // check if we should run the intermediate solution callback this iteration
    if(i >= cb_iters.front() || is_converged || is_stopped) {
      backend_->current_solution(cur_primal_sol_, cur_dual_sol_);
 
      if(opts_.verbose) {
        int digits = std::floor(std::log10( (double) opts_.max_iters )) + 1;
        cout << "It " << std::setw(digits) << (i + 1) << ": " << std::scientific;
        cout.precision(2);
        cout << "Feas_p=" << primal_res;
        cout << ", Eps_p=" << eps_pri;
        cout << ", Feas_d=" << dual_res;
        cout << ", Eps_d=" << eps_dua << "; ";
      }

      // MATLAB callback
      interm_cb_(i + 1, cur_primal_sol_, cur_dual_sol_);
      
      cb_iters.pop_front();
    }

    if(is_stopped) {
      if(opts_.verbose)
        std::cout << "Stopped by user." << std::endl;

      return Solver<T>::ConvergenceResult::kStoppedUser;
    }

    if(is_converged) {
      if(opts_.verbose)
        std::cout << "Reached convergence tolerance." << std::endl;

      return Solver<T>::ConvergenceResult::kConverged;
    }
  }

  if(opts_.verbose)
    std::cout << "Reached maximum iterations." << std::endl;

  return Solver<T>::ConvergenceResult::kStoppedMaxIters;
}

template<typename T>
void Solver<T>::Release() {
  problem_->Release();
  backend_->Release();
}

// Explicit template instantiation
template class Solver<float>;
template class Solver<double>;

} // namespace prost
