#include "hip/hip_runtime.h"
#include "solver_backend_pdhg.hpp"

#include <hip/hip_complex.h> // TODO: what was this needed for ...?
#include <iostream>
#include <sstream>

#include "util/cuwrap.hpp"

/**
 * @brief ...
 */
__global__
void ComputeBtNumeratorPDHG(
    real *d_res_dual,
    real *d_kx,
    real *d_kx_prev,
    real *d_y,
    real *d_y_prev,
    real *d_left,
    real *d_right,
    int m)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx >= m)
    return;

  const real diff_y = d_y[idx] - d_y_prev[idx]; // / d_left is not needed?
  const real diff_kx = d_kx[idx] - d_kx_prev[idx];
  d_res_dual[idx] = diff_y * diff_kx;
}

/**
 * @brief ...
 */
__global__
void ComputeBtDenom1PDHG(
    real *d_res_primal,
    real *d_x,
    real *d_x_prev,
    real *d_right,
    int n)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx >= n)
    return;

  const real diff = (d_x[idx] - d_x_prev[idx]);
  d_res_primal[idx] = diff * diff / d_right[idx];
}

/**
 * @brief ...
 */
__global__
void ComputeBtDenom2PDHG(
    real *d_res_dual,
    real *d_y,
    real *d_y_prev,
    real *d_left,
    int m)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx >= m)
    return;

  const real diff = (d_y[idx] - d_y_prev[idx]);
  d_res_dual[idx] = diff * diff / d_left[idx];
}

/**
 * @brief ...
 */
__global__
void ComputeProxArgPrimalPDHG(
    real *d_prox_arg,
    real *d_x,
    real tau,
    real *d_right,
    real *d_kty,
    int n) {
  
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx >= n)
    return;

  d_prox_arg[idx] = d_x[idx] - tau * d_right[idx] * d_kty[idx];
}

/**
 * @brief ...
 */
__global__
void ComputeProxArgDualPDHG(
    real *d_prox_arg,
    real *d_y,
    real sigma,
    real theta,
    real *d_left,
    real *d_kx,
    real *d_kx_prev,
    int m) {
  
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx >= m)
    return;

  d_prox_arg[idx] = d_y[idx] + sigma * d_left[idx] *
      ((1 + theta) * d_kx[idx] - theta * d_kx_prev[idx]);
}

/**
 * @brief ...
 */
__global__
void ComputePrimalResidualPDHG(
    real *d_res_primal,
    real *d_x,
    real *d_x_prev,
    real *d_kty,
    real *d_kty_prev,
    real tau,
    real *d_right,
    int n)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx >= n)
    return;

  d_res_primal[idx] =
      ((d_x_prev[idx] - d_x[idx]) / (tau * d_right[idx])) -
      (d_kty_prev[idx] - d_kty[idx]);
}

/**
 * @brief ...
 */
__global__
void ComputeDualResidualPDHG(
    real *d_res_dual,
    real *d_y,
    real *d_y_prev,
    real *d_kx,
    real *d_kx_prev,
    real sigma,
    real *d_left,
    real theta,
    int m)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx >= m)
    return;

  // TODO: derive residual for theta!=1?
  d_res_dual[idx] =
      ((d_y_prev[idx] - d_y[idx]) / (sigma * d_left[idx])) -
      (d_kx_prev[idx] - d_kx[idx]);
}

void SolverBackendPDHG::PerformIteration() {
  int n = problem_.mat->ncols();
  int m = problem_.mat->nrows();
  
  dim3 block(kBlockSizeCUDA, 1, 1);
  dim3 grid_n((n + block.x - 1) / block.x, 1, 1);
  dim3 grid_m((m + block.x - 1) / block.x, 1, 1);

  // gradient descent step
  ComputeProxArgPrimalPDHG<<<grid_n, block>>>(
      d_prox_arg_,
      d_x_,
      tau_,
      problem_.precond->right(),
      d_kty_,
      n);
  hipDeviceSynchronize();

  // remember previous primal iterate
  std::swap(d_x_, d_x_prev_);

  // apply prox_g
  for(int j = 0; j < problem_.prox_g.size(); ++j)
    problem_.prox_g[j]->Evaluate(
        d_prox_arg_,
        d_x_,
        tau_,
        problem_.precond->right());

  // compute Kx^{k+1} and remember Kx^k
  std::swap(d_kx_, d_kx_prev_);
  problem_.mat->MultVec(d_x_, d_kx_, false, 1, 0);

  // gradient ascent step
  ComputeProxArgDualPDHG<<<grid_m, block>>>
      (d_prox_arg_,
       d_y_,
       sigma_,
       theta_,
       problem_.precond->left(),
       d_kx_,
       d_kx_prev_,
       m);
  hipDeviceSynchronize();

  // apply prox_hc
  std::swap(d_y_, d_y_prev_);
  for(int j = 0; j < problem_.prox_hc.size(); ++j)
    problem_.prox_hc[j]->Evaluate(
        d_prox_arg_,
        d_y_,
        sigma_,
        problem_.precond->left());

  // compute K^T y^{k+1} and remember Ky^k
  std::swap(d_kty_, d_kty_prev_);
  problem_.mat->MultVec(d_y_, d_kty_, true, 1, 0);
  
  // compute residuals
  ComputePrimalResidualPDHG<<<grid_n, block>>>(
      d_res_primal_,
      d_x_,
      d_x_prev_,
      d_kty_,
      d_kty_prev_,
      tau_,
      problem_.precond->right(),
      n);

  ComputeDualResidualPDHG<<<grid_m, block>>>(
      d_res_dual_,
      d_y_,
      d_y_prev_,
      d_kx_,
      d_kx_prev_,
      sigma_,
      problem_.precond->left(),
      theta_,
      m);
  
  hipDeviceSynchronize();

  cuwrap::asum<real>(cublas_handle_, d_res_primal_, n, &res_primal_);
  cuwrap::asum<real>(cublas_handle_, d_res_dual_, m, &res_dual_);

  //std::cout << res_primal_ << "," << res_dual_ << std::endl;

  // if backtracking is enabled, update step sizes
  if(opts_.pdhg == kPDHGBacktrack) {
    real num, denom1, denom2;
    
    // compute numerator
    ComputeBtNumeratorPDHG<<<grid_m, block>>>(
        d_res_dual_,
        d_kx_,
        d_kx_prev_,
        d_y_,
        d_y_prev_,
        problem_.precond->left(),
        problem_.precond->right(),
        m);
    hipDeviceSynchronize();
    cuwrap::asum<real>(cublas_handle_, d_res_dual_, m, &num);
    
    // compute denominator
    ComputeBtDenom1PDHG<<<grid_n, block>>>(
        d_res_primal_,
        d_x_,
        d_x_prev_,
        problem_.precond->right(),
        n);
    hipDeviceSynchronize();
    cuwrap::asum<real>(cublas_handle_, d_res_primal_, n, &denom1);

    ComputeBtDenom2PDHG<<<grid_m, block>>>(
        d_res_dual_,
        d_y_,
        d_y_prev_,
        problem_.precond->left(),
        m);
    hipDeviceSynchronize();
    cuwrap::asum<real>(cublas_handle_, d_res_dual_, m, &denom2);

    real b = (2.0 * tau_ * sigma_ * num) / (opts_.bt_gamma * (sigma_ * denom1 + tau_ * denom2));

    if(b > 1) {
      std::cout << "bt_gamma=" << opts_.bt_gamma << std::endl;
      std::cout << "num=" << num << ", denom1=" << denom1 << ", denom2=" << denom2 << std::endl;
      std::cout << b << ", " << tau_ << ", " << sigma_ << ", tau*sigma=" << tau_ * sigma_ << std::endl;

      tau_ = opts_.bt_beta * tau_ / b;
      sigma_ = opts_.bt_beta * sigma_ / b;

      std::cout << "new_tau=" << tau_ << ", new_sigma=" << sigma_ << ", " << tau_ * sigma_ << std::endl;
    }
  }
  
  // adapt step-sizes according to chosen algorithm
  switch(opts_.pdhg) {
    case kPDHGAlg1: // fixed step sizes, do nothing.
      break;

    case kPDHGAlg2: // adapt based on strong convexity constant gamma
      // TODO: implement me!
      break;

    case kPDHGBacktrack: 
    case kPDHGAdapt: { // adapt based on residuals

      if(res_primal_ > opts_.s * res_dual_ * opts_.delta) {
        tau_ = tau_ / (1 - alpha_);
        sigma_ = sigma_ * (1 - alpha_);
        alpha_ = alpha_ * opts_.nu;
      }
      if(res_primal_ < opts_.s * res_dual_ / opts_.delta) {
        tau_ = tau_ * (1 - alpha_);
        sigma_ = sigma_ / (1 - alpha_);
        alpha_ = alpha_ * opts_.nu;
      }

    } break;
  }

  //std::cout << res_primal_ << ", " << res_dual_ << std::endl;
}

bool SolverBackendPDHG::Initialize() {
  int m = problem_.mat->nrows();
  int n = problem_.mat->ncols();
  int l = std::max(m, n);

  hipMalloc((void **)&d_x_, n * sizeof(real));
  hipMalloc((void **)&d_x_prev_, n * sizeof(real));
  hipMalloc((void **)&d_kty_, n * sizeof(real));
  hipMalloc((void **)&d_kty_prev_, n * sizeof(real));
  hipMalloc((void **)&d_res_primal_, n * sizeof(real));
  hipMalloc((void **)&d_y_, m * sizeof(real));
  hipMalloc((void **)&d_y_prev_, m * sizeof(real));
  hipMalloc((void **)&d_kx_, m * sizeof(real));
  hipMalloc((void **)&d_kx_prev_, m * sizeof(real));
  hipMalloc((void **)&d_res_dual_, m * sizeof(real));
  hipMalloc((void **)&d_prox_arg_, l * sizeof(real));  

  tau_ = 1;
  sigma_ = 1;
  theta_ = 1;
  alpha_ = opts_.alpha0;

  // TODO: add possibility for non-zero initializations
  hipMemset(d_x_, 0, n * sizeof(real));
  hipMemset(d_x_prev_, 0, n * sizeof(real));
  hipMemset(d_kty_, 0, n * sizeof(real));
  hipMemset(d_kty_prev_, 0, n * sizeof(real));
  hipMemset(d_res_primal_, 0, n * sizeof(real));
  hipMemset(d_y_, 0, m * sizeof(real));
  hipMemset(d_y_prev_, 0, m * sizeof(real));
  hipMemset(d_res_dual_, 0, m * sizeof(real));
  hipMemset(d_kx_, 0, m * sizeof(real));
  hipMemset(d_kx_prev_, 0, m * sizeof(real));
  hipMemset(d_prox_arg_, 0, l * sizeof(real));

  hipblasCreate(&cublas_handle_);
  
  return true;
}

void SolverBackendPDHG::Release() {
  hipblasDestroy(cublas_handle_);

  hipFree(d_x_);
  hipFree(d_y_);
  hipFree(d_x_prev_);
  hipFree(d_y_prev_);
  hipFree(d_prox_arg_);
  hipFree(d_kx_);
  hipFree(d_kty_);
  hipFree(d_kx_prev_);
  hipFree(d_kty_prev_);
  hipFree(d_res_primal_);
  hipFree(d_res_dual_);
}

void SolverBackendPDHG::iterates(real *primal, real *dual) {
  hipMemcpy(primal, d_x_, sizeof(real) * problem_.mat->ncols(), hipMemcpyDeviceToHost);
  hipMemcpy(dual, d_y_, sizeof(real) * problem_.mat->nrows(), hipMemcpyDeviceToHost);
}

bool SolverBackendPDHG::converged() {
  return false; //std::max(res_primal_, res_dual_) < opts_.tolerance;
}

std::string SolverBackendPDHG::status() {
  std::stringstream ss;

  return ss.str();
}

int SolverBackendPDHG::gpu_mem_amount() {
  int m = problem_.mat->nrows();
  int n = problem_.mat->ncols();

  return (5 * (n + m) + std::max(n, m)) * sizeof(real);
}
