#include "hip/hip_runtime.h"
#include "linop/linop_data_prec.hpp"
#include <iostream>

template<typename T>
LinOpDataPrec<T>::LinOpDataPrec(size_t row,
                                size_t col,
                                size_t nx,
                                size_t ny,
                                size_t L,
                                T left, T right)
    : LinOp<T>(row, col, nx*ny*(L + 1 + 2*(L-1)), nx*ny*(L + 2*(L-1))), nx_(nx), ny_(ny), L_(L), left_(left), right_(left)
{
}

template<typename T>
__global__
void LinOpDataPrecKernel(T *d_res,
                           T *d_rhs,
                           size_t nx,
                           size_t ny,
                           size_t L,
                           T left,
                           T right)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t y = y_tilde  % ny;
  size_t l = y_tilde / ny;

  if(x >= nx || y >= ny || l >= L)
    return;

  size_t idx_u = y + x * ny + l * nx * ny;
  size_t idx_s = nx*ny*L + L*ny*x + y*L + l;
  size_t idx_w = nx*ny*L + (L-1)*ny*nx + L*ny*x + y*L + l;
  


  T delta_t = (right - left) / (L-1);
  T t = left + l * delta_t;
 

  d_res[idx_u] += d_rhs[idx_u];
  if(l==0) {
    d_res[idx_u] += (1 / delta_t) * (d_rhs[idx_s] + (t+delta_t) * d_rhs[idx_w]);    
  } else if(l==L) {
    d_res[idx_u] += 
        (1 / delta_t) * (-d_rhs[idx_s-1] - 
        (t-delta_t) * d_rhs[idx_w-1]);   
  } else {
    d_res[idx_u] += 
        (1 / delta_t) * (-d_rhs[idx_s-1] + d_rhs[idx_s] - 
        (t-delta_t) * d_rhs[idx_w-1] + (t+delta_t) * d_rhs[idx_w]);
  }
}


template<typename T>
__global__
void LinOpDataPrecAdjointUKernel(T *d_res,
                             T *d_rhs,
                             size_t nx,
                             size_t ny,
                             size_t L, T left, T right)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t y = y_tilde  % ny;
  size_t l = y_tilde / ny;

  if(x >= nx || y >= ny || l >= L)
    return;

  size_t idx = y + x * ny + l * nx * ny;

  d_res[idx] += d_rhs[idx];
}

template<typename T>
__global__
void LinOpDataPrecAdjointSKernel(T *d_res,
                             T *d_rhs,
                             size_t nx,
                             size_t ny,
                             size_t L, T left, T right)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t y = y_tilde  % ny;
  size_t l = y_tilde / ny;

  if(x >= nx || y >= ny || l >= L-1)
    return;

  size_t idx_v = y + x * ny + l * nx * ny;
  size_t idx_v1 = y + x * ny + (l+1) * nx * ny;
  size_t idx_s = nx*ny*L + x*ny*L + y*L + l;
  size_t idx_r = idx_s + nx*ny;
  
  T delta_t = (right - left) / (L-1);
  
  d_res[idx_s] += d_rhs[idx_r] + (1 / delta_t) * (d_rhs[idx_v] - d_rhs[idx_v1]);
}

template<typename T>
__global__
void LinOpDataPrecAdjointWKernel(T *d_res,
                             T *d_rhs,
                             size_t nx,
                             size_t ny,
                             size_t L, T left, T right)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t y = y_tilde  % ny;
  size_t l = y_tilde / ny;

  if(x >= nx || y >= ny || l >= L-1)
    return;

  size_t idx_v = y + x * ny + l * nx * ny;
  size_t idx_v1 = y + x * ny + (l+1) * nx * ny;
  size_t idx_w = nx*ny*L + nx*ny*(L-1) + x*ny*L + y*L + l;
  size_t idx_z = idx_w + nx*ny;
  size_t idx_q = nx*ny*L + x*ny + y;
  
  T delta_t = (right - left) / (L-1);
  T t = left + l * delta_t;
  
  d_res[idx_w] += -d_rhs[idx_q] + d_rhs[idx_z] + (1 / delta_t) * ((t+delta_t) * d_rhs[idx_v] - t*d_rhs[idx_v1]);
}

template<typename T>
LinOpDataPrec<T>::~LinOpDataPrec() {
}

template<typename T>
void LinOpDataPrec<T>::EvalLocalAdd(T *d_res, T *d_rhs) {

  dim3 block(1, 128, 1);
  dim3 grid((nx_ + block.x - 1) / block.x,
            (ny_*L_ + block.y - 1) / block.y,
            1);

  LinOpDataPrecKernel<<<grid, block>>>(d_res, d_rhs, nx_, ny_, L_, left_, right_);
}

template<typename T>
void LinOpDataPrec<T>::EvalAdjointLocalAdd(T *d_res, T *d_rhs) {

  dim3 block(2, 128, 1);
  dim3 gridU((nx_ + block.x - 1) / block.x,
            (ny_*(L_-1) + block.y - 1) / block.y,
            1);
  dim3 gridSW((nx_ + block.x - 1) / block.x,
            (ny_*(L_-1) + block.y - 1) / block.y,
            1);

  LinOpDataPrecAdjointUKernel<T><<<gridU, block>>>(d_res, d_rhs, nx_, ny_, L_, left_, right_);
  LinOpDataPrecAdjointSKernel<T><<<gridSW, block>>>(d_res, d_rhs, nx_, ny_, L_-1, left_, right_);
  LinOpDataPrecAdjointWKernel<T><<<gridSW, block>>>(d_res, d_rhs, nx_, ny_, L_-1, left_, right_);
}

template<typename T>
T LinOpDataPrec<T>::row_sum(size_t row, T alpha) const {
    if(row >= nx_*ny_*L_)
        return 1;

    size_t l = row % L_;

    T delta_t = (right_ - left_) / (L_-1);
    T t = left_ + l * delta_t;


    if(l == 0)
        return 1 + (1 + t + delta_t) / delta_t;
    if(l==L_)
        return 1 + (1 + t - delta_t) / delta_t;

    return 1 + (2*(1+t)) / delta_t;
}
  
template<typename T>
T LinOpDataPrec<T>::col_sum(size_t col, T alpha) const {
    if(col < nx_*ny_*L_)
        return 1;

    T delta_t = (right_ - left_) / (L_-1);
    if(col < nx_*ny_*L_ + nx_*ny_*(L_-1))
        return 1 + (2 / delta_t);

    size_t l = col % (L_-1);
    T t = left_ + l * delta_t;

    return 2 + (t + t + delta_t) / delta_t;
}

template class LinOpDataPrec<float>;
template class LinOpDataPrec<double>;
