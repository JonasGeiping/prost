#include "hip/hip_runtime.h"
/**
* This file is part of prost.
*
* Copyright 2016 Thomas Möllenhoff <thomas dot moellenhoff at in dot tum dot de> 
* and Emanuel Laude <emanuel dot laude at in dot tum dot de> (Technical University of Munich)
*
* prost is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* prost is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with prost. If not, see <http://www.gnu.org/licenses/>.
*/

#include <thrust/transform_reduce.h>
#include <iostream>
#include <sstream>

#include "prost/linop/block_sparse.hpp"
#include "prost/exception.hpp"

namespace prost {

template<> hipsparseHandle_t BlockSparse<float>::cusp_handle_ = nullptr;
template<> hipsparseHandle_t BlockSparse<double>::cusp_handle_ = nullptr;

template<typename T>
BlockSparse<T>* BlockSparse<T>::CreateFromCSC(
  size_t row,
  size_t col,
  int m,
  int n,
  int nnz,
  const std::vector<T>& val,
  const std::vector<int32_t>& ptr,
  const std::vector<int32_t>& ind)
{
  BlockSparse<T> *block = new BlockSparse<T>(row, col, m, n);
  block->nnz_ = nnz;

  // create data on host
  block->host_ind_t_ = ind; 
  block->host_ptr_t_ = ptr; 
  block->host_val_t_ = val; 

  block->host_ind_.resize(block->nnz_);
  block->host_val_.resize(block->nnz_);
  block->host_ptr_.resize(block->nrows() + 1);

  csr2csc(
    block->ncols(), 
    block->nrows(), 
    block->nnz_, 
    &block->host_val_t_[0],
    &block->host_ind_t_[0],
    &block->host_ptr_t_[0],
    &block->host_val_[0],
    &block->host_ind_[0],
    &block->host_ptr_[0]);

  return block;
}

template<typename T>
BlockSparse<T>::BlockSparse(size_t row, size_t col, size_t nrows, size_t ncols)
  : Block<T>(row, col, nrows, ncols)
{
}

template<typename T>
BlockSparse<T>::~BlockSparse()
{
}

template<typename T>
void BlockSparse<T>::Initialize()
{
  if(cusp_handle_ == nullptr)
    hipsparseCreate(&cusp_handle_);

  hipsparseCreateMatDescr(&descr_);
  hipsparseSetMatType(descr_, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr_, HIPSPARSE_INDEX_BASE_ZERO);

  // forward
  ind_.resize(nnz_);
  val_.resize(nnz_);
  ptr_.resize(this->nrows() + 1);

  // transpose
  ind_t_.resize(nnz_);
  val_t_.resize(nnz_);
  ptr_t_.resize(this->ncols() + 1);

  // copy to GPU
  thrust::copy(host_ind_t_.begin(), host_ind_t_.end(), ind_t_.begin());
  thrust::copy(host_ptr_t_.begin(), host_ptr_t_.end(), ptr_t_.begin());
  thrust::copy(host_val_t_.begin(), host_val_t_.end(), val_t_.begin());

  thrust::copy(host_ind_.begin(), host_ind_.end(), ind_.begin());
  thrust::copy(host_ptr_.begin(), host_ptr_.end(), ptr_.begin());
  thrust::copy(host_val_.begin(), host_val_.end(), val_.begin());
}

template<typename T>
T BlockSparse<T>::row_sum(size_t row, T alpha) const
{
  T sum = 0;

  for(int32_t i = host_ptr_[row]; i < host_ptr_[row + 1]; i++)
    sum += std::pow(std::abs(host_val_[i]), alpha);

  return sum;
}

template<typename T>
T BlockSparse<T>::col_sum(size_t col, T alpha) const
{
  T sum = 0;

  for(int32_t i = host_ptr_t_[col]; i < host_ptr_t_[col + 1]; i++)
    sum += std::pow(std::abs(host_val_t_[i]), alpha);

  return sum;
}

template<typename T>
size_t BlockSparse<T>::gpu_mem_amount() const
{
  size_t total_bytes = 0;

  total_bytes += 2 * nnz_ * sizeof(int32_t);
  total_bytes += (this->nrows() + this->ncols() + 2) * sizeof(int32_t);
  total_bytes += 2 * nnz_ * sizeof(T);

  return total_bytes;
}

template<>
void BlockSparse<float>::EvalLocalAdd(
  const typename thrust::device_vector<float>::iterator& res_begin,
  const typename thrust::device_vector<float>::iterator& res_end,
  const typename thrust::device_vector<float>::const_iterator& rhs_begin,
  const typename thrust::device_vector<float>::const_iterator& rhs_end)
{
  hipsparseStatus_t stat;
  const float alpha = 1;
  const float beta = 1;

  stat = hipsparseScsrmv(cusp_handle_,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    nrows(),
    ncols(),
    nnz_,
    &alpha,
    descr_,
    thrust::raw_pointer_cast(val_.data()),
    thrust::raw_pointer_cast(ptr_.data()),
    thrust::raw_pointer_cast(ind_.data()),
    thrust::raw_pointer_cast(&(*rhs_begin)),
    &beta,
    thrust::raw_pointer_cast(&(*res_begin)));

  if(stat != HIPSPARSE_STATUS_SUCCESS)
  {
    std::ostringstream ss;
    ss << "Sparse Matrix-Vector multiplication failed. Error code = " << stat << ".";

    throw Exception(ss.str());
  }
}

template<>
void BlockSparse<float>::EvalAdjointLocalAdd(
  const typename thrust::device_vector<float>::iterator& res_begin,
  const typename thrust::device_vector<float>::iterator& res_end,
  const typename thrust::device_vector<float>::const_iterator& rhs_begin,
  const typename thrust::device_vector<float>::const_iterator& rhs_end)
{
  hipsparseStatus_t stat;
  const float alpha = 1;
  const float beta = 1;

  stat = hipsparseScsrmv(cusp_handle_,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    ncols(),
    nrows(),
    nnz_,
    &alpha,
    descr_,
    thrust::raw_pointer_cast(val_t_.data()),
    thrust::raw_pointer_cast(ptr_t_.data()),
    thrust::raw_pointer_cast(ind_t_.data()),
    thrust::raw_pointer_cast(&(*rhs_begin)),
    &beta,
    thrust::raw_pointer_cast(&(*res_begin)));

  if(stat != HIPSPARSE_STATUS_SUCCESS)
  {
    std::ostringstream ss;
    ss << "Sparse Matrix-Vector multiplication failed. Error code = " << stat << ".";

    throw Exception(ss.str());
  }
}

template<>
void BlockSparse<double>::EvalLocalAdd(
  const typename thrust::device_vector<double>::iterator& res_begin,
  const typename thrust::device_vector<double>::iterator& res_end,
  const typename thrust::device_vector<double>::const_iterator& rhs_begin,
  const typename thrust::device_vector<double>::const_iterator& rhs_end)
{
  hipsparseStatus_t stat;
  const double alpha = 1;
  const double beta = 1;

  stat = hipsparseDcsrmv(cusp_handle_,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    nrows(),
    ncols(),
    nnz_,
    &alpha,
    descr_,
    thrust::raw_pointer_cast(val_.data()),
    thrust::raw_pointer_cast(ptr_.data()),
    thrust::raw_pointer_cast(ind_.data()),
    thrust::raw_pointer_cast(&(*rhs_begin)),
    &beta,
    thrust::raw_pointer_cast(&(*res_begin)));

  if(stat != HIPSPARSE_STATUS_SUCCESS)
  {
    std::ostringstream ss;
    ss << "Sparse Matrix-Vector multiplication failed. Error code = " << stat << ".";

    throw Exception(ss.str());
  }
}

template<>
void BlockSparse<double>::EvalAdjointLocalAdd(
  const typename thrust::device_vector<double>::iterator& res_begin,
  const typename thrust::device_vector<double>::iterator& res_end,
  const typename thrust::device_vector<double>::const_iterator& rhs_begin,
  const typename thrust::device_vector<double>::const_iterator& rhs_end)
{
  hipsparseStatus_t stat;
  const double alpha = 1;
  const double beta = 1;

  stat = hipsparseDcsrmv(cusp_handle_,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    ncols(),
    nrows(),
    nnz_,
    &alpha,
    descr_,
    thrust::raw_pointer_cast(val_t_.data()),
    thrust::raw_pointer_cast(ptr_t_.data()),
    thrust::raw_pointer_cast(ind_t_.data()),
    thrust::raw_pointer_cast(&(*rhs_begin)),
    &beta,
    thrust::raw_pointer_cast(&(*res_begin)));

  if(stat != HIPSPARSE_STATUS_SUCCESS)
  {
    std::ostringstream ss;
    ss << "Sparse Matrix-Vector multiplication failed. Error code = " << stat << ".";

    throw Exception(ss.str());
  }
}

// Explicit template instantiation
template class BlockSparse<float>;
template class BlockSparse<double>;

} // namespace prost