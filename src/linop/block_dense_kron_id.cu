#include "hip/hip_runtime.h"
/**
* This file is part of prost.
*
* Copyright 2016 Thomas Möllenhoff <thomas dot moellenhoff at in dot tum dot de> 
* and Emanuel Laude <emanuel dot laude at in dot tum dot de> (Technical University of Munich)
*
* prost is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* prost is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with prost. If not, see <http://www.gnu.org/licenses/>.
*/

#include "prost/linop/block_dense_kron_id.hpp"
#include "prost/config.hpp"
#include "prost/exception.hpp"

namespace prost
{

template<typename T, bool transpose>
__global__ void BlockDenseKronIdKernel(
    T *result,
    const T *rhs,
    size_t diaglength,
    size_t nrows,
    size_t ncols,
    const T *data)
{
  const size_t tx = threadIdx.x + blockIdx.x * blockDim.x;

  if(!transpose) {
    if(tx < diaglength * nrows) {
      size_t row = tx / diaglength;
      size_t col_ofs = tx % diaglength;

      T sum = 0;
      for(int32_t i = 0; i < ncols; i++) {
	sum += data[i * nrows + row] * rhs[i * diaglength + col_ofs];
      }

      result[tx] += sum;
    }
  }
  else {
    if(tx < diaglength * ncols) {
      size_t col = tx / diaglength;
      size_t row_ofs = tx % diaglength;

      T sum = 0;
      for(int32_t i = 0; i < nrows; i++) {
	sum += data[i + col * nrows] * rhs[i * diaglength + row_ofs];
      }

      result[tx] += sum;
    }
  }
}

template<typename T>
BlockDenseKronId<T>::BlockDenseKronId(size_t row, size_t col, size_t nrows, size_t ncols)
    : Block<T>(row, col, nrows, ncols)
{
}

template<typename T>
BlockDenseKronId<T> *BlockDenseKronId<T>::CreateFromColFirstData(
    size_t diaglength,
    size_t row,
    size_t col,
    size_t nrows,
    size_t ncols,
    const std::vector<T>& data)
{
  BlockDenseKronId<T> *block = new BlockDenseKronId<T>(row, col, ((size_t)nrows) * diaglength, ((size_t)ncols) * diaglength);

  block->diaglength_ = diaglength;
  block->mat_nrows_ = nrows;
  block->mat_ncols_ = ncols;
  block->host_data_ = data;

  return block;
}

template<typename T>
void BlockDenseKronId<T>::Initialize()
{
  data_.resize(this->mat_nrows_ * this->mat_ncols_);
  thrust::copy(host_data_.begin(), host_data_.end(), data_.begin());
}

template<typename T>
T BlockDenseKronId<T>::row_sum(size_t row, T alpha) const
{
  row = row / diaglength_;
  
  T sum = 0;
  for(int32_t i = 0; i < mat_ncols_; i++)
    sum += std::pow(std::abs(host_data_[i * mat_nrows_ + row]), alpha);

  return sum;
}

template<typename T>
T BlockDenseKronId<T>::col_sum(size_t col, T alpha) const
{
  col = col / diaglength_;
  
  T sum = 0;
  for(int32_t i = 0; i < mat_nrows_; i++)
    sum += std::pow(std::abs(host_data_[i + col * mat_nrows_]), alpha);

  return sum;
}

template<typename T>
size_t BlockDenseKronId<T>::gpu_mem_amount() const
{
  return host_data_.size() * sizeof(T);
}

template<typename T>
void BlockDenseKronId<T>::EvalLocalAdd(
    const typename device_vector<T>::iterator& res_begin,
    const typename device_vector<T>::iterator& res_end,
    const typename device_vector<T>::const_iterator& rhs_begin,
    const typename device_vector<T>::const_iterator& rhs_end)
{
  dim3 block(kBlockSizeCUDA, 1, 1);
  dim3 grid((this->nrows() + block.x) / block.x, 1, 1);

  BlockDenseKronIdKernel<T, false>
      <<<grid, block>>>(
          thrust::raw_pointer_cast(&(*res_begin)),
          thrust::raw_pointer_cast(&(*rhs_begin)),
          diaglength_,
          mat_nrows_,
          mat_ncols_,
          thrust::raw_pointer_cast(data_.data()));
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and throw exception
    std::stringstream ss;
    ss << "BlockDenseKronId (forward): CUDA error: " << hipGetErrorString(error) << std::endl;
    throw Exception(ss.str());
  }
}

template<typename T>
void BlockDenseKronId<T>::EvalAdjointLocalAdd(
    const typename device_vector<T>::iterator& res_begin,
    const typename device_vector<T>::iterator& res_end,
    const typename device_vector<T>::const_iterator& rhs_begin,
    const typename device_vector<T>::const_iterator& rhs_end)
{
  dim3 block(kBlockSizeCUDA, 1, 1);
  dim3 grid((this->ncols() + block.x) / block.x, 1, 1);

  BlockDenseKronIdKernel<T, true>
      <<<grid, block>>>(
          thrust::raw_pointer_cast(&(*res_begin)),
          thrust::raw_pointer_cast(&(*rhs_begin)),
          diaglength_,
          mat_nrows_,
          mat_ncols_,
          thrust::raw_pointer_cast(data_.data()));
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and throw exception
    std::stringstream ss;
    ss << "BlockDenseKronId (adjoint): CUDA error: " << hipGetErrorString(error) << std::endl;
    throw Exception(ss.str());
  }
}

// Explicit template instantiation
template class BlockDenseKronId<float>;
template class BlockDenseKronId<double>;

} // namespace prost