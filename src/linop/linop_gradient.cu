#include "hip/hip_runtime.h"
#include "linop_gradient.hpp"

template<typename T>
__global__
void LinOpGradient2DKernel(T *d_rhs,
                           T *d_res,
                           size_t nx,
                           size_t ny,
                           size_t L)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y = threadIdx.y + blockDim.y * blockIdx.y;
  size_t L = threadIdx.z + blockDim.z * blockIdx.z;

  T gx, gy;
  size_t idx = L + y * L + x * ny * L;

  if(x < nx - 1)
    gx = d_rhs[idx + ny * L] - d_rhs[idx];
  else
    gx = static_cast<T>(0);

  if(y < ny - 1)
    gy = d_rhs[idx + y * L] - d_rhs[idx];
  else
    gy = static_cast<T>(0);

  d_res[y + x * ny + L * (ny * nx)] += gx;
  d_res[y + x * ny + L * (ny * nx) + nx * ny * L] += gy;
}


template<typename T>
__global__
void LinOpDivergence2DKernel(T *d_rhs,
                             T *d_res,
                             size_t nx,
                             size_t ny,
                             size_t L)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y = threadIdx.y + blockDim.y * blockIdx.y;
  size_t L = threadIdx.z + blockDim.z * blockIdx.z;

  T divx, divy;
  size_t idx = L + y * L + x * ny * L;

  divx = d_rhs[idx];
  divy = d_rhs[idx + nx * ny * L];

  if(x > 0)
    divx -= d_rhs[idx - ny * L];

  if(y > 0)
    divy -= d_rhs[idx - y * L];

  d_res[idx] -= (divx + divy); // adjoint is minus the divergence
}

template<typename T>
__global__
void LinOpGradient3DKernel(T *d_rhs,
                           T *d_res,
                           size_t nx,
                           size_t ny,
                           size_t L)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y = threadIdx.y + blockDim.y * blockIdx.y;
  size_t L = threadIdx.z + blockDim.z * blockIdx.z;

  T gx, gy, gl;
  size_t idx = L + y * L + x * ny * L;

  if(x < nx - 1)
    gx = d_rhs[idx + ny * L] - d_rhs[idx];
  else
    gx = static_cast<T>(0);

  if(y < ny - 1)
    gy = d_rhs[idx + y * L] - d_rhs[idx];
  else
    gy = static_cast<T>(0);

  if(L < L - 1)
    gl = d_rhs[idx + 1] - d_rhs[idx];
  else
    gl = static_cast<T>(0);

  d_res[y + x * ny + L * (ny * nx)] += gx;
  d_res[y + x * ny + L * (ny * nx) + nx * ny * L] += gy;
  d_res[y + x * ny + L * (ny * nx) + 2 * nx * ny * L] += gl;
}


template<typename T>
__global__
void LinOpDivergence3DKernel(T *d_rhs,
                             T *d_res,
                             size_t nx,
                             size_t ny,
                             size_t L)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y = threadIdx.y + blockDim.y * blockIdx.y;
  size_t L = threadIdx.z + blockDim.z * blockIdx.z;

  T divx, divy, divl;
  size_t idx = L + y * L + x * ny * L;

  divx = d_rhs[idx];
  divy = d_rhs[idx + nx * ny * L];
  divl = d_rhs[idx + 2 * nx * ny * L];

  if(x > 0)
    divx -= d_rhs[idx - ny * L];

  if(y > 0)
    divy -= d_rhs[idx + nx * ny * L - y * L];

  if(l > 0)
    divl -= d_rhs[idx + 2 * nx * ny * L - 1];

  d_res[idx] -= (divx + divy + divl); // adjoint is minus the divergence
}

template<typename T>
LinOpGradient2D<T>::LinOpGradient2D(size_t row, size_t col, size_t nx, size_t ny, size_t L)
    : nx_(nx), ny_(ny), L_(L)
{
  this->row_ = row;
  this->col_ = col;
  this->nrows_ = nx*ny*L*2;
  this->ncols_ = nx*ny*L; 
}

template<typename T>
LinOpGradient2D<T>::LinOpGradient2D() {
}

template<typename T>
void LinOpGradient2D<T>::EvalLocalAdd(T *d_res, T *d_rhs) {

  dim3 block(16, 16, 8);
  dim3 grid((nx + block.x - 1) / block.x,
            (ny + block.y - 1) / block.y,
            (L + block.z - 1) / block.z);

  LinOpGradientKernel<<<grid, block>>>(d_rhs, d_res, nx_, ny_, L_);
}

template<typename T>
void LinOpGradient2D<T>::EvalAdjointLocalAdd(T *d_res, T *d_rhs) {
  
  dim3 block(16, 16, 8);
  dim3 grid((nx + block.x - 1) / block.x,
            (ny + block.y - 1) / block.y,
            (L + block.z - 1) / block.z);

  LinOpMinusDivergenceKernel<<<grid, block>>>(d_rhs, d_res, nx_, ny_, L_);
}

template<typename T>
LinOpGradient3D<T>::LinOpGradient3D(size_t row, size_t col, size_t nx, size_t ny, size_t L)
    : nx_(nx), ny_(ny), L_(L)
{
  this->row_ = row;
  this->col_ = col;
  this->nrows_ = nx*ny*L*3;
  this->ncols_ = nx*ny*L; 
}

template<typename T>
LinOpGradient3D<T>::LinOpGradient3D() {
}

template<typename T>
void LinOpGradient3D<T>::EvalLocalAdd(T *d_res, T *d_rhs) {

  dim3 block(16, 16, 8);
  dim3 grid((nx + block.x - 1) / block.x,
            (ny + block.y - 1) / block.y,
            (L + block.z - 1) / block.z);

  LinOpGradientKernel<<<grid, block>>>(d_rhs, d_res, nx_, ny_, L_);
}

template<typename T>
void LinOpGradient3D<T>::EvalAdjointLocalAdd(T *d_res, T *d_rhs) {
  
  dim3 block(16, 16, 8);
  dim3 grid((nx + block.x - 1) / block.x,
            (ny + block.y - 1) / block.y,
            (L + block.z - 1) / block.z);

  LinOpMinusDivergenceKernel<<<grid, block>>>(d_rhs, d_res, nx_, ny_, L_);
}

template class LinOpGradient2D<float>;
template class LinOpGradient2D<double>;
template class LinOpGradient3D<float>;
template class LinOpGradient3D<double>;
