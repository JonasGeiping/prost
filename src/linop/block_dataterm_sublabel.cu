#include "hip/hip_runtime.h"
#include "prost/linop/block_dataterm_sublabel.hpp"

namespace prost {

template<typename T>
__global__
void BlockDatatermSublabelKernel(T *d_res,
				 const T *d_rhs,
				 size_t nx,
				 size_t ny,
				 size_t L,
				 T t_min,
				 T t_max)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t l = y_tilde % (L-1);
  size_t y = y_tilde / (L-1);
  
  if(x >= nx || y >= ny || l >= L-1)
    return;
  
  size_t idx = (L-1)*ny*x + y*(L-1) + l;
  T delta_t = (t_max - t_min) / (L-1);
  T t = t_min + l * delta_t;
  T add_v = -t*d_rhs[idx];
  
  for(size_t i = 1; i < L-1-l; i++)
  {
    add_v += delta_t*d_rhs[idx+i];
  }

  d_res[idx] += add_v;
}

template<typename T>
__global__
void BlockDatatermSublabelAdjointKernel(T *d_res,
					const T *d_rhs,
					size_t nx,
					size_t ny,
					size_t L,
					T t_min,
					T t_max)
{
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y_tilde = threadIdx.y + blockDim.y * blockIdx.y;
  size_t l = y_tilde % (L-1);
  size_t y = y_tilde / (L-1);

  if(x >= nx || y >= ny || l >= L-1)
    return;

  size_t idx = (L-1)*ny*x + y*(L-1) + l;
  T delta_t = (t_max - t_min) / (L-1);
  T t = t_min + l * delta_t;
  T add_s = -t*d_rhs[idx];

  for(size_t i = 1; i <= l; i++)
    add_s += delta_t*d_rhs[idx-i];

  d_res[idx] += add_s;
}
  
template<typename T>
BlockDatatermSublabel<T>::BlockDatatermSublabel(size_t row, 
						size_t col, 
						size_t nx, 
						size_t ny, 
						size_t L, 
						T left, 
						T right)
  : Block<T>(row,col,nx*ny*(L-1),nx*ny*(L-1)), nx_(nx), ny_(ny), L_(L), t_min_(left), t_max_(right)
{
}

template<typename T>
T BlockDatatermSublabel<T>::row_sum(size_t row, T alpha) const
{
  size_t l = row % (L_-1);
  T delta_t = (t_max_ - t_min_) / (L_-1);
  T t = t_min_ + l * delta_t;
  return t + (L_-2-l)*delta_t;  
}

template<typename T>
T BlockDatatermSublabel<T>::col_sum(size_t col, T alpha) const
{
  size_t l = col % (L_-1);
  T delta_t = (t_max_ - t_min_) / (L_-1);
  T t = t_min_ + l * delta_t;

  return t + l*delta_t;
}

template<typename T>
void BlockDatatermSublabel<T>::EvalLocalAdd(
    const typename device_vector<T>::iterator& res_begin,
    const typename device_vector<T>::iterator& res_end,
    const typename device_vector<T>::const_iterator& rhs_begin,
    const typename device_vector<T>::const_iterator& rhs_end)
{
  dim3 block(1, 128, 1);
  dim3 grid((nx_ + block.x - 1) / block.x,
            (ny_*(L_-1) + block.y - 1) / block.y,
            1);

  BlockDatatermSublabelKernel<T>
    <<<grid, block>>>(thrust::raw_pointer_cast(&(*res_begin)),
		      thrust::raw_pointer_cast(&(*rhs_begin)),
		      nx_,
		      ny_,
		      L_,
		      t_min_,
		      t_max_);
}

template<typename T>
void BlockDatatermSublabel<T>::EvalAdjointLocalAdd(
    const typename device_vector<T>::iterator& res_begin,
    const typename device_vector<T>::iterator& res_end,
    const typename device_vector<T>::const_iterator& rhs_begin,
    const typename device_vector<T>::const_iterator& rhs_end)
{
  dim3 block(1, 128, 1);
  dim3 grid((nx_ + block.x - 1) / block.x,
            (ny_*(L_-1) + block.y - 1) / block.y,
            1);

  BlockDatatermSublabelAdjointKernel<T>
    <<<grid, block>>>(thrust::raw_pointer_cast(&(*res_begin)),
		      thrust::raw_pointer_cast(&(*rhs_begin)),
		      nx_,
		      ny_,
		      L_,
		      t_min_,
		      t_max_);
}

// Explicit template instantiation
template class BlockDatatermSublabel<float>;
template class BlockDatatermSublabel<double>;
  
} // namespace prost
